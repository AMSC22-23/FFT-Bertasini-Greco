#include "hip/hip_runtime.h"
#include "IterativeFastFourierCUDA.cuh"
#include "bitreverse.hpp"

#include <hip/hip_runtime.h>
#include <cuda/std/complex>
#include <cuda/std/cmath>
#include <iostream>
#include <cmath>


// typedef cuda::std::complex<double> cpx;
using cpx = cuda::std::complex<double>;

__global__ void fft_kernel(cpx *x, int N, int m, int is_inverse) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < N / m) {
        int k_m = k * m;
        cpx Wm = cuda::std::polar(1.0, (1-2*is_inverse)*-2*M_PI/m);
        cpx W = 1;
        for (int j = 0; j < m/2; j++) {
            cpx t = W * x[k_m + j + m/2];
            cpx u = x[k_m + j];
            x[k_m + j] = u + t;
            x[k_m + j + m/2] = u - t;
            W *= Wm;
        }
    }
}

auto fftCU (Typedefs::vcpx& x, const bool is_inverse) -> void{
    size_t N = x.size();
    if (N == 1) return;
    
    // Bit reverse copy
    bit_reverse_copy(x);

    cpx *d_x;
    hipMalloc((void**)&d_x, N * sizeof(cpx));
    hipMemcpy(d_x, x.data(), N * sizeof(cpx), hipMemcpyHostToDevice);

    int blockSize = 256;
    for (size_t s = 1; s <= log2(N); s++) {
        size_t m = 1 << s;
        int gridSize = (N / m + blockSize - 1) / blockSize;
        fft_kernel<<<gridSize, blockSize>>>(d_x, N, m, is_inverse);
        hipDeviceSynchronize();
    }

    hipMemcpy(x.data(), d_x, N * sizeof(cpx), hipMemcpyDeviceToHost);
    hipFree(d_x);
}