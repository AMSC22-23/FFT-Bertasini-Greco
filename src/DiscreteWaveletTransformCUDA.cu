#include "hip/hip_runtime.h"
#include "DiscreteWaveletTransformCUDA.cuh"

//CUDA implementation
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <hip/hip_runtime.h>

namespace Typedefs {
    using vec = std::vector<double>;
}

#define CUDA_CALL(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << "@" << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void transformKernel(double* signal, const double* t_mat, const double* temp, int sub_step, int sub_size, const size_t matrix_size) {
    int j = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    if (j >= sub_size) return;
    int index_signal = j*sub_step;
    signal[index_signal] = 0;
    signal[index_signal+sub_step] = 0;
    for (unsigned long m=0; m < matrix_size; m+=1){
        signal[index_signal]              += temp[j+m]*t_mat[m];
        signal[index_signal+sub_step]     += temp[j+m]*t_mat[m+ matrix_size];
    }
}

auto cudabackend::dwtCU(Typedefs::vec &signal, const bool is_inverse, const std::span<const Typedefs::DType> &transform_matrix, const std::span<const Typedefs::DType> &inverse_matrix, const uint8_t user_levels) -> void
{
    auto& t_mat = is_inverse ? inverse_matrix : transform_matrix;
    const unsigned long matrix_size = t_mat.size() / 2;

    std::vector<double> temp;
    int levels = user_levels == 0 ? log2(signal.size()) : user_levels;
    int start = is_inverse ? levels-1 : 0; 
    int end = is_inverse ? -1 : levels;
    int step = is_inverse ? -1 : 1;

    double* d_signal;
    double* d_t_mat;
    double* d_temp;

    CUDA_CALL(hipMalloc((void**)&d_signal, signal.size() * sizeof(double)));
    CUDA_CALL(hipMalloc((void**)&d_t_mat, t_mat.size() * sizeof(double)));

    CUDA_CALL(hipMemcpy(d_signal, signal.data(), signal.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_t_mat, t_mat.data(), t_mat.size() * sizeof(double), hipMemcpyHostToDevice));

    for (int i = start; i != end; i += step) {
        temp.clear();
        int sub_step = pow(2, i);
        int sub_size = signal.size()/sub_step;

        for (int j = 0; j < sub_size; j++) temp.push_back(signal[j*sub_step]);
    
        if (!is_inverse) for (unsigned long j = 0; j < matrix_size-2; j++) temp.push_back(temp[j]);
        else             for (unsigned long j = 0; j < matrix_size-2; j++) temp.insert(temp.begin(), *(temp.end()-1-j));

        CUDA_CALL(hipMalloc((void**)&d_temp, temp.size() * sizeof(double)));
        CUDA_CALL(hipMemcpy(d_temp, temp.data(), temp.size() * sizeof(double), hipMemcpyHostToDevice));

        int threadsPerBlock = 256;
        int blocksPerGrid = (sub_size + threadsPerBlock - 1) / threadsPerBlock;
        transformKernel<<<blocksPerGrid, threadsPerBlock>>>(d_signal, d_t_mat, d_temp, sub_step, sub_size, matrix_size);
        CUDA_CALL(hipDeviceSynchronize());
        CUDA_CALL(hipFree(d_temp));
        CUDA_CALL(hipMemcpy(signal.data(), d_signal, signal.size() * sizeof(double), hipMemcpyDeviceToHost));
    }

    CUDA_CALL(hipMemcpy(signal.data(), d_signal, signal.size() * sizeof(double), hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_signal));
    CUDA_CALL(hipFree(d_t_mat));
}