#include "hip/hip_runtime.h"
#include "DiscreteWaveletTransformCUDA.cuh"

//CUDA implementation
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <hip/hip_runtime.h>

namespace Typedefs {
    using vec = std::vector<double>;
}

#define CUDA_CALL(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << "@" << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

template <unsigned long matrix_size>
__global__ void transformKernel(double* signal, const double* t_mat, const double* temp, int sub_step, int sub_size) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= sub_size / 2) return;

    int index_signal = j * 2 * sub_step;
    signal[index_signal] = 0;
    signal[index_signal + sub_step] = 0;
    for (unsigned long m = 0; m < matrix_size*2; ++m) {
        signal[index_signal] += temp[j * 2 + m] * t_mat[m];
        signal[index_signal + sub_step] += temp[j * 2 + m] * t_mat[m + matrix_size*2];
    }
}

template <unsigned long matrix_size>
auto dwtCU(Typedefs::vec &signal, bool is_inverse, const std::array<double, matrix_size*2> &transform_matrix, const std::array<double, matrix_size*2> &inverse_matrix, const int user_levels) -> void {
    const auto& t_mat = is_inverse ? inverse_matrix : transform_matrix;
    int levels = user_levels == 0 ? log2(signal.size()) : user_levels;
    int start = is_inverse ? levels - 1 : 0;
    int end = is_inverse ? -1 : levels;
    int step = is_inverse ? -1 : 1;

    double* d_signal;
    double* d_t_mat;
    double* d_temp;
    size_t signal_size = signal.size() * sizeof(double);
    size_t t_mat_size = matrix_size * 2 * sizeof(double);  // Since t_mat contains two matrix_size parts

    CUDA_CALL(hipMalloc((void**)&d_signal, signal_size));
    CUDA_CALL(hipMalloc((void**)&d_t_mat, t_mat_size));

    CUDA_CALL(hipMemcpy(d_signal, signal.data(), signal_size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_t_mat, t_mat.data(), t_mat_size, hipMemcpyHostToDevice));

    for (int i = start; i != end; i += step) {
        int sub_step = pow(2, i);
        int sub_size = signal.size() / sub_step;
        std::vector<double> temp;

        for (int j = 0; j < sub_size; j++) temp.push_back(signal[j * sub_step]);

        if (!is_inverse) {
            for (unsigned long j = 0; j < matrix_size*2 - 2; j++) temp.push_back(temp[j]);
        } else {
            for (unsigned long j = 0; j < matrix_size*2 - 2; j++) temp.insert(temp.begin(), *(temp.end() - 1 - j));
        }

        CUDA_CALL(hipMalloc((void**)&d_temp, temp.size() * sizeof(double)));
        CUDA_CALL(hipMemcpy(d_temp, temp.data(), temp.size() * sizeof(double), hipMemcpyHostToDevice));

        int threadsPerBlock = 256;
        int blocksPerGrid = (sub_size / 2 + threadsPerBlock - 1) / threadsPerBlock;
        transformKernel<matrix_size><<<blocksPerGrid, threadsPerBlock>>>(d_signal, d_t_mat, d_temp, sub_step, sub_size);
        CUDA_CALL(hipDeviceSynchronize());
    }

    CUDA_CALL(hipMemcpy(signal.data(), d_signal, signal_size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(d_signal));
    CUDA_CALL(hipFree(d_t_mat));
    CUDA_CALL(hipFree(d_temp));
}
    

template void dwtCU<2> (Typedefs::vec&, bool, const std::array<double, 4>&, const std::array<double, 4>&, const int);
template void dwtCU<4> (Typedefs::vec&, bool, const std::array<double, 8>&, const std::array<double, 8>&, const int);
template void dwtCU<6> (Typedefs::vec&, bool, const std::array<double, 12>&, const std::array<double, 12>&, const int);
template void dwtCU<8> (Typedefs::vec&, bool, const std::array<double, 16>&, const std::array<double, 16>&, const int);
template void dwtCU<10>(Typedefs::vec&, bool, const std::array<double, 20>&, const std::array<double, 20>&, const int);
template void dwtCU<16>(Typedefs::vec&, bool, const std::array<double, 32>&, const std::array<double, 32>&, const int);
template void dwtCU<20>(Typedefs::vec&, bool, const std::array<double, 40>&, const std::array<double, 40>&, const int);
template void dwtCU<30>(Typedefs::vec&, bool, const std::array<double, 60>&, const std::array<double, 60>&, const int);
template void dwtCU<40>(Typedefs::vec&, bool, const std::array<double, 80>&, const std::array<double, 80>&, const int);
